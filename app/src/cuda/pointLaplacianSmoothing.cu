#include "hip/hip_runtime.h"
﻿#include "cudaUtils.cuh"
#include <stdio.h>

#define NEIGHBORINDEXSTEP 1

__global__ void planePointsLaplacianSmoothing_kernel(
    float* inputPoints,float* outputPoints, unsigned int w, unsigned int h)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int index = y * w + x;

    for (int i = 0; i < ATTRIBUTESIZE; i++) {
        outputPoints[index * ATTRIBUTESIZE + ATTRIBUTE_OFFSET_VERTEX + i] = inputPoints[index * ATTRIBUTESIZE + ATTRIBUTE_OFFSET_VERTEX + i];
    }
    // self is valid point
    if (inputPoints[index * ATTRIBUTESIZE + ATTRIBUTE_OFFSET_VERTEX + 2] != 0) {
        float coordinateSum[3] = { 0,0,0 };
        int validNeighborCount = 0;
        for (int shiftY = -NEIGHBORINDEXSTEP; shiftY <= NEIGHBORINDEXSTEP; shiftY++) {
            for (int shiftX = -NEIGHBORINDEXSTEP; shiftX <= NEIGHBORINDEXSTEP; shiftX++) {

                if ((y + shiftY) > 0 &&
                    (y + shiftY) < h &&
                    (shiftX + x) > 0 &&
                    (shiftX + x) < w
                    )
                {
                    int indexNeighbor = (y + shiftY) * w + (shiftX + x);

                    // neighbor is valid point
                    if (inputPoints[indexNeighbor * ATTRIBUTESIZE + ATTRIBUTE_OFFSET_VERTEX + 2] != 0) {
                        coordinateSum[0] += inputPoints[indexNeighbor * ATTRIBUTESIZE + ATTRIBUTE_OFFSET_VERTEX + 0];
                        coordinateSum[1] += inputPoints[indexNeighbor * ATTRIBUTESIZE + ATTRIBUTE_OFFSET_VERTEX + 1];
                        coordinateSum[2] += inputPoints[indexNeighbor * ATTRIBUTESIZE + ATTRIBUTE_OFFSET_VERTEX + 2];
                        validNeighborCount++;
                    }
                }
            }
        }
        if (validNeighborCount != 0) {
            outputPoints[index * ATTRIBUTESIZE + ATTRIBUTE_OFFSET_VERTEX + 0] = coordinateSum[0] / validNeighborCount;
            outputPoints[index * ATTRIBUTESIZE + ATTRIBUTE_OFFSET_VERTEX + 1] = coordinateSum[1] / validNeighborCount;
            outputPoints[index * ATTRIBUTESIZE + ATTRIBUTE_OFFSET_VERTEX + 2] = coordinateSum[2] / validNeighborCount;
        }
    }    
}

void launch_kernel(float* pos, unsigned int w, unsigned int h,int iteration)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(w / block.x, h / block.y, 1);
    {
        bool dstIstmp = true;
        float* tmpPointsArray;
        hipMalloc((void**)&tmpPointsArray, w * h * ATTRIBUTESIZE * sizeof(float));

        // dilation
        for (int i = 0; i < iteration; i++) {
            if (dstIstmp) {
                planePointsLaplacianSmoothing_kernel << < grid, block >> > (pos, tmpPointsArray, w, h);
            }
            else {
                planePointsLaplacianSmoothing_kernel << < grid, block >> > (tmpPointsArray, pos, w, h);
            }
            dstIstmp = !dstIstmp;
        }

        if (!dstIstmp) {
            hipMemcpy(tmpPointsArray, pos, w * h * ATTRIBUTESIZE * sizeof(uint16_t), hipMemcpyDeviceToDevice);
        }
        hipFree(tmpPointsArray);
    }
}

void CudaAlogrithm::planePointsLaplacianSmoothing(struct hipGraphicsResource** vbo_resource,
    unsigned int w, unsigned int h, int interation
)
{
    // map OpenGL buffer object for writing from CUDA
    float* dptr;
    hipGraphicsMapResources(1, vbo_resource, 0);
    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void**)&dptr, &num_bytes, *vbo_resource);

    launch_kernel(dptr, w, h, interation);

    // unmap buffer object
    hipGraphicsUnmapResources(1, vbo_resource, 0);
}