#include "hip/hip_runtime.h"
﻿#include "cudaUtils.cuh"
#include <stdio.h>

__global__ void depthVisualize_kernel(
    hipSurfaceObject_t mask, uint16_t* depthRaw, hipSurfaceObject_t output, unsigned int w, unsigned int h,float depthScale, float far)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int index = y * w + x;

    uchar4 pixelCenter = { 0,0,0,0 };
    surf2Dread(&pixelCenter, mask, x * sizeof(uchar4), y);
    float depth = float(depthRaw[index]) * depthScale / far * 255;
    bool isCulled = (depth > 255) || (pixelCenter.w == 0);

    uchar4 pixel = {
        isCulled ? 255:depth,
        isCulled ? 0 : depth,
        isCulled ? 0 : depth,
        1.0
    };
    //Write the new pixel color to the 
    surf2Dwrite(pixel, output, x * sizeof(uchar4), y);
}

void launch_kernel(
    hipSurfaceObject_t mask, uint16_t* depthRaw, hipSurfaceObject_t output, unsigned int w, unsigned int h, float depthScale, float far)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(w / block.x, h / block.y, 1);
    depthVisualize_kernel << < grid, block >> > (mask, depthRaw, output, w, h, depthScale, far);
}

void CudaAlogrithm::depthVisualize(
    hipGraphicsResource_t*maskTexture, hipGraphicsResource_t* cudaTexture,
    uint16_t* depthRaw, unsigned int w, unsigned int h, float depthScale, float far)
{
    hipArray* texture_ptr;

    hipGraphicsMapResources(1, cudaTexture, 0);
    hipGraphicsSubResourceGetMappedArray(&texture_ptr, *cudaTexture, 0, 0);

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = texture_ptr;
    hipSurfaceObject_t surfObject;
    hipCreateSurfaceObject(&surfObject, &resDesc);

    hipArray* mask_ptr;

    hipGraphicsMapResources(1, maskTexture, 0);
    hipGraphicsSubResourceGetMappedArray(&mask_ptr, *maskTexture, 0, 0);

    hipResourceDesc resDesc2;
    memset(&resDesc2, 0, sizeof(resDesc2));
    resDesc2.resType = hipResourceTypeArray;
    resDesc2.res.array.array = mask_ptr;
    hipSurfaceObject_t surfObject2;
    hipCreateSurfaceObject(&surfObject2, &resDesc2);
    // You now have a CUDA Surface object that refers to the GL texture.
    // Write to the Surface using CUDA.
    launch_kernel(surfObject2, depthRaw, surfObject, w, h, depthScale, far);

    // We're not going to use this Surface object again.  We'll make a new one next frame.
    hipDestroySurfaceObject(surfObject);
    hipGraphicsUnmapResources(1, cudaTexture, 0);
}