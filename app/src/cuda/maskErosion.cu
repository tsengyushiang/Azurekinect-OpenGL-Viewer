#include "hip/hip_runtime.h"
﻿#include "cudaUtils.cuh"
#include <stdio.h>

__global__ void maskErosion_kernel(
    hipSurfaceObject_t input, hipSurfaceObject_t output,
    unsigned int w, unsigned int h, int erosionPixel)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    uchar4 pixelCenter = { 0,0,0,0 };
    surf2Dread(&pixelCenter, input, x * sizeof(uchar4), y);

    if (pixelCenter.w > 0) {
        for (int shiftX = -erosionPixel; shiftX <= erosionPixel; shiftX++) {
            for (int shiftY = -erosionPixel; shiftY <= erosionPixel; shiftY++) {

                if (
                    (y + shiftY) > 0 &&
                    (y + shiftY) < h &&
                    (shiftX + x) > 0 &&
                    (shiftX + x) < w
                    ) 
                {

                    uchar4 pixelNeighbor = { 0,0,0,0 };
                    surf2Dread(&pixelNeighbor, input, (shiftX + x) * sizeof(uchar4), (y + shiftY));

                    if (pixelNeighbor.w == 0) {
                        pixelCenter.w = 0;
                        surf2Dwrite(pixelCenter, output, x * sizeof(uchar4), y);
                        return;
                    }
                }
            }
        }
    }
}

void launch_kernel(hipSurfaceObject_t input,hipSurfaceObject_t output, unsigned int w, unsigned int h, int erosionPixel)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(w / block.x, h / block.y, 1);
    maskErosion_kernel << < grid, block >> > (input,output, w, h, erosionPixel);
}

void CudaAlogrithm::maskErosion(hipGraphicsResource_t* cudaTexture, unsigned int w, unsigned int h, int erosionPixel)
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    hipArray* cuInputArray;
    hipMallocArray(&cuInputArray, &channelDesc, w, h, hipArraySurfaceLoadStore);

    hipArray* texture_ptr;

    hipGraphicsMapResources(1, cudaTexture, 0);
    hipGraphicsSubResourceGetMappedArray(&texture_ptr, *cudaTexture, 0, 0);

    cudaMemcpyArrayToArray(cuInputArray, 0, 0, texture_ptr, 0,0, w * h * sizeof(uchar4));

    hipResourceDesc origion;
    memset(&origion, 0, sizeof(origion));
    origion.resType = hipResourceTypeArray;
    origion.res.array.array = texture_ptr;
    hipSurfaceObject_t surfObject;
    hipCreateSurfaceObject(&surfObject, &origion);

    hipResourceDesc copyinput;
    memset(&copyinput, 0, sizeof(copyinput));
    copyinput.resType = hipResourceTypeArray;
    copyinput.res.array.array = cuInputArray;
    hipSurfaceObject_t surfObjectCopy;
    hipCreateSurfaceObject(&surfObjectCopy, &copyinput);

    // You now have a CUDA Surface object that refers to the GL texture.
    // Write to the Surface using CUDA.
    launch_kernel(surfObjectCopy,surfObject, w, h, erosionPixel);

    // We're not going to use this Surface object again.  We'll make a new one next frame.
    hipDestroySurfaceObject(surfObject);
    hipDestroySurfaceObject(surfObjectCopy);
    hipFreeArray(cuInputArray);
    hipGraphicsUnmapResources(1, cudaTexture, 0);
}