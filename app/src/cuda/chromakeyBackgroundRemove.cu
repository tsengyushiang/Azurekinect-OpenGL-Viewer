#include "hip/hip_runtime.h"
#include "cudaUtils.cuh"
#include <stdio.h>

__global__ void chromaKeyBackgroundRemove_kernel(
    unsigned char* colorRaw, hipSurfaceObject_t output, unsigned int w, unsigned int h, glm::vec3 chromakey,float threshold)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int index = y * w + x;

    glm::vec3 color = glm::vec3(
        colorRaw[index * 3 + 2],
        colorRaw[index * 3 + 1],
        colorRaw[index * 3 + 0]
    );

    uchar4 pixel = {
        colorRaw[index * 3 + 2],
        colorRaw[index * 3 + 1],
        colorRaw[index * 3 + 0],
        glm::distance(color,chromakey)> threshold ?255:0
    };
    //Write the new pixel color to the 
    surf2Dwrite(pixel, output, x * sizeof(uchar4), y);
}

void launch_kernel(
    unsigned char* colorRaw, hipSurfaceObject_t output, unsigned int mesh_width, unsigned int mesh_height, glm::vec3 color, float threshold)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    chromaKeyBackgroundRemove_kernel << < grid, block >> > (colorRaw, output, mesh_width, mesh_height, color, threshold);
}

void CudaAlogrithm::chromaKeyBackgroundRemove(
    hipGraphicsResource_t* cudaTexture,
    unsigned char* colorRaw, unsigned int w, unsigned int h,glm::vec3 color, float threshold)
{
    hipArray* texture_ptr;

    hipGraphicsMapResources(1, cudaTexture, 0);
    hipGraphicsSubResourceGetMappedArray(&texture_ptr, *cudaTexture, 0, 0);


    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = texture_ptr;
    hipSurfaceObject_t surfObject;
    hipCreateSurfaceObject(&surfObject, &resDesc);
    // You now have a CUDA Surface object that refers to the GL texture.
    // Write to the Surface using CUDA.
    launch_kernel(colorRaw, surfObject, w, h, color, threshold);

    // We're not going to use this Surface object again.  We'll make a new one next frame.
    hipDestroySurfaceObject(surfObject);
    hipGraphicsUnmapResources(1, cudaTexture, 0);
}