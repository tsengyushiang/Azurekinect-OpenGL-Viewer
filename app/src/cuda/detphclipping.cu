#include "hip/hip_runtime.h"
﻿#include "cudaUtils.cuh"
#include <stdio.h>

__global__ void clipFloorAndFarDepth_kernel(
    hipSurfaceObject_t mask, uint16_t* depthRaw, unsigned int w, unsigned int h,float* xy_table, float depthScale,float farplane, 
    glm::vec3 planeCenter, glm::vec3 planeNormal,float planeCullingDistance
)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int index = (h - 1 - y) * w + x;

    uchar4 pixelCenter = { 0,0,0,0 };
    surf2Dread(&pixelCenter, mask, x * sizeof(uchar4), y);

    float depthValue = (float)depthRaw[index] * depthScale;
    if (!ISVALIDDEPTHVALUE(depthValue)) {
        pixelCenter.w = 0; 
    }

    glm::vec4 localPos3d = glm::vec4(
        xy_table[index * 2] * depthValue,
        xy_table[index * 2+1] * depthValue,
        depthValue,
        1.0
    );

    if (depthValue > farplane) {
        pixelCenter.w = 0;
    }

    glm::vec3 planeCoordinate = glm::vec3(localPos3d.x, localPos3d.y, localPos3d.z) - planeCenter;

    double distance2plane = (glm::dot<3, float, glm::qualifier::highp>(planeCoordinate, planeNormal))/ glm::length(planeNormal);

    if (depthValue!=0 && distance2plane > planeCullingDistance) {
        pixelCenter.w = 0;
    }

    surf2Dwrite(pixelCenter, mask, x * sizeof(uchar4), y);
}

void launch_kernel(
    hipSurfaceObject_t mask, uint16_t* depthRaw, 
    unsigned int w, unsigned int h,float* xy_table, float depthScale, float farplane,
    glm::vec3 planeCenter, glm::vec3 planeNormal, float planeCullingDistance
)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(w / block.x, h / block.y, 1);
    clipFloorAndFarDepth_kernel << < grid, block >> > (mask, depthRaw, w, h, xy_table, depthScale, farplane, planeCenter, planeNormal, planeCullingDistance);
}

void CudaAlogrithm::clipFloorAndFarDepth(
    hipGraphicsResource_t* maskTexture,
    uint16_t* depthRaw, unsigned int w, unsigned int h, float* xy_table, float depthScale, float farplane,
    glm::vec3 planeCenter, glm::vec3 planeNormal, float planeCullingDistance
)
{
    hipArray* mask_ptr;

    hipGraphicsMapResources(1, maskTexture, 0);
    hipGraphicsSubResourceGetMappedArray(&mask_ptr, *maskTexture, 0, 0);

    hipResourceDesc resDesc2;
    memset(&resDesc2, 0, sizeof(resDesc2));
    resDesc2.resType = hipResourceTypeArray;
    resDesc2.res.array.array = mask_ptr;
    hipSurfaceObject_t surfObject2;
    hipCreateSurfaceObject(&surfObject2, &resDesc2);
    // You now have a CUDA Surface object that refers to the GL texture.
    // Write to the Surface using CUDA.
    launch_kernel(surfObject2, depthRaw, w, h, xy_table, depthScale, farplane, planeCenter, planeNormal, planeCullingDistance);

    // We're not going to use this Surface object again.  We'll make a new one next frame.
    hipDestroySurfaceObject(surfObject2);
    hipGraphicsUnmapResources(1, maskTexture, 0);
}